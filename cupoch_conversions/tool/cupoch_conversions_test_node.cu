// prj hdrs
#include "cupoch_conversions/cupoch_conversions.h"

// ros hdrs
#include <ros/ros.h>
#include <sensor_msgs/PointCloud2.h>
#include <std_msgs/Int32.h>

using namespace std;
using namespace cupoch;

std::string camera_point_topic;
auto cloud = std::make_shared<geometry::PointCloud>();
sensor_msgs::PointCloud2 m_pub_cupoch_pc;

ros::Publisher time_pub;

void points_callback(const sensor_msgs::PointCloud2ConstPtr& points_msg)
{
    ROS_INFO_STREAM_ONCE("points_callback");

    auto t1 = ros::WallTime::now();
    auto t2 = ros::WallTime::now();

    t1 = ros::WallTime::now();
    cupoch_conversions::rosToCupoch(points_msg, cloud);
    t2 = ros::WallTime::now();
    std_msgs::Int32 time;
    time.data = (t2 - t1).toSec() * 1000.0;
    time_pub.publish(time);
    ROS_INFO_STREAM("rosToCupoch processing_time: " << (t2 - t1).toSec() * 1000.0 << "[ms]");

    if (cloud->HasPoints())
    {
        ROS_INFO("this msg has %d points", cloud->points_.size());
    }

    ROS_INFO_STREAM_ONCE("points_callback end");
}

int main(int argc, char** argv)
{
    ros::init(argc, argv, "cupoch_conversions_test_node");
    ros::NodeHandle private_nh("~");

    utility::SetVerbosityLevel(utility::VerbosityLevel::Debug);

#ifndef __aarch64__
    utility::InitializeAllocator(utility::rmmAllocationMode_t::PoolAllocation, 1000000000);
    hipSetDeviceFlags( hipDeviceScheduleBlockingSync);
#else
    // use managed memory allocation to speed up memcpy
    utility::InitializeAllocator(utility::rmmAllocationMode_t::CudaManagedMemory, 1000000000);
#endif

    private_nh.param("camera3d_point_topic", camera_point_topic, std::string("/points_cloud"));

    auto points_sub = private_nh.subscribe(camera_point_topic, 10, points_callback);
    time_pub = private_nh.advertise<std_msgs::Int32>("/cupoch/ros2cupoch", 1);

    ros::spin();

    return 0;
}