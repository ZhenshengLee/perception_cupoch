// Copyright 2020 Autonomous Robots Lab, University of Nevada, Reno

// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at

//     http://www.apache.org/licenses/LICENSE-2.0

// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include "cupoch_conversions/cupoch_conversions.h"
#include "cupoch/utility/platform.h"

using namespace cupoch;

namespace cupoch_conversions
{
  // d2h: get points
  void cupochToRos(std::shared_ptr<cupoch::geometry::PointCloud> &pointcloud, sensor_msgs::PointCloud2 &ros_pc2, std::string frame_id)
  {
    // d2h
    // hipMemcpy to improve speed, Async to lower the cpu usage
    thrust::host_vector<Eigen::Vector3f> pointcloud_points_host;
    pointcloud_points_host.resize(pointcloud->points_.size());
    thrust::host_vector<Eigen::Vector3f> pointcloud_colors_host;
    pointcloud_colors_host.resize(pointcloud->points_.size());

    hipStream_t s0 = utility::GetStream(0);
    hipStream_t s1 = utility::GetStream(1);
    hipStreamAttachMemAsync(s0, thrust::raw_pointer_cast(pointcloud->points_.data()));
    cudaSafeCall(hipMemcpyAsync(pointcloud_points_host.data(), thrust::raw_pointer_cast(pointcloud->points_.data()),
                            pointcloud->points_.size() * sizeof(Eigen::Vector3f), hipMemcpyDeviceToHost, s0));
    hipStreamAttachMemAsync(s1, thrust::raw_pointer_cast(pointcloud->colors_.data()));
    cudaSafeCall(hipMemcpyAsync(pointcloud_colors_host.data(), thrust::raw_pointer_cast(pointcloud->colors_.data()),
                            pointcloud->colors_.size() * sizeof(Eigen::Vector3f), hipMemcpyDeviceToHost, s1));
    hipDeviceSynchronize();

    sensor_msgs::PointCloud2Modifier modifier(ros_pc2);
    if (pointcloud->HasColors())
    {
      modifier.setPointCloud2FieldsByString(2, "xyz", "rgb");
    }
    else
    {
      modifier.setPointCloud2FieldsByString(1, "xyz");
    }
    modifier.resize(pointcloud_points_host.size());
    ros_pc2.header.frame_id = frame_id;
    sensor_msgs::PointCloud2Iterator<float> ros_pc2_x(ros_pc2, "x");
    sensor_msgs::PointCloud2Iterator<float> ros_pc2_y(ros_pc2, "y");
    sensor_msgs::PointCloud2Iterator<float> ros_pc2_z(ros_pc2, "z");

    if (pointcloud->HasColors())
    {
      sensor_msgs::PointCloud2Iterator<uint8_t> ros_pc2_r(ros_pc2, "r");
      sensor_msgs::PointCloud2Iterator<uint8_t> ros_pc2_g(ros_pc2, "g");
      sensor_msgs::PointCloud2Iterator<uint8_t> ros_pc2_b(ros_pc2, "b");
      for (size_t i = 0; i < pointcloud_points_host.size();
           i++, ++ros_pc2_x, ++ros_pc2_y, ++ros_pc2_z, ++ros_pc2_r, ++ros_pc2_g, ++ros_pc2_b)
      {
        const Eigen::Vector3f &point = pointcloud_points_host[i];
        const Eigen::Vector3f &color = pointcloud_colors_host[i];
        *ros_pc2_x = point(0);
        *ros_pc2_y = point(1);
        *ros_pc2_z = point(2);
        *ros_pc2_r = (uint8_t)(255 * color(0));
        *ros_pc2_g = (uint8_t)(255 * color(1));
        *ros_pc2_b = (uint8_t)(255 * color(2));
      }
    }
    else
    {
      for (size_t i = 0; i < pointcloud_points_host.size(); i++, ++ros_pc2_x, ++ros_pc2_y, ++ros_pc2_z)
      {
        const Eigen::Vector3f &point = pointcloud_points_host[i];
        *ros_pc2_x = point(0);
        *ros_pc2_y = point(1);
        *ros_pc2_z = point(2);
      }
    }
  }

  // gpu不能随意push_back
  // h2d
  void rosToCupoch(const sensor_msgs::PointCloud2ConstPtr &ros_pc2, std::shared_ptr<cupoch::geometry::PointCloud> &cupoch_pc, bool skip_colors)
  {
    // host
    thrust::host_vector<Eigen::Vector3f> cupoch_pc_points_host;
    thrust::host_vector<Eigen::Vector3f> cupoch_pc_colors_host;

    sensor_msgs::PointCloud2ConstIterator<float> ros_pc2_x(*ros_pc2, "x");
    sensor_msgs::PointCloud2ConstIterator<float> ros_pc2_y(*ros_pc2, "y");
    sensor_msgs::PointCloud2ConstIterator<float> ros_pc2_z(*ros_pc2, "z");
    cupoch_pc_points_host.reserve(ros_pc2->height * ros_pc2->width);
    if (ros_pc2->fields.size() == 3 || skip_colors == true)
    {
      for (size_t i = 0; i < ros_pc2->height * ros_pc2->width; ++i, ++ros_pc2_x, ++ros_pc2_y, ++ros_pc2_z)
      {
        cupoch_pc_points_host.push_back(Eigen::Vector3f(*ros_pc2_x, *ros_pc2_y, *ros_pc2_z));
      }
    }
    else
    {
      cupoch_pc_colors_host.reserve(ros_pc2->height * ros_pc2->width);
      if (ros_pc2->fields[3].name == "rgb")
      {
        sensor_msgs::PointCloud2ConstIterator<uint8_t> ros_pc2_r(*ros_pc2, "r");
        sensor_msgs::PointCloud2ConstIterator<uint8_t> ros_pc2_g(*ros_pc2, "g");
        sensor_msgs::PointCloud2ConstIterator<uint8_t> ros_pc2_b(*ros_pc2, "b");

        for (size_t i = 0; i < ros_pc2->height * ros_pc2->width;
             ++i, ++ros_pc2_x, ++ros_pc2_y, ++ros_pc2_z, ++ros_pc2_r, ++ros_pc2_g, ++ros_pc2_b)
        {
          cupoch_pc_points_host.push_back(Eigen::Vector3f(*ros_pc2_x, *ros_pc2_y, *ros_pc2_z));
          cupoch_pc_colors_host.push_back(Eigen::Vector3f(((int)(*ros_pc2_r)) / 255.0, ((int)(*ros_pc2_g)) / 255.0,
                                                   ((int)(*ros_pc2_b)) / 255.0));
        }
      }
      else if (ros_pc2->fields[3].name == "intensity")
      {
        sensor_msgs::PointCloud2ConstIterator<uint8_t> ros_pc2_i(*ros_pc2, "intensity");
        for (size_t i = 0; i < ros_pc2->height * ros_pc2->width;
             ++i, ++ros_pc2_x, ++ros_pc2_y, ++ros_pc2_z, ++ros_pc2_i)
        {
          cupoch_pc_points_host.push_back(Eigen::Vector3f(*ros_pc2_x, *ros_pc2_y, *ros_pc2_z));
          cupoch_pc_colors_host.push_back(Eigen::Vector3f(*ros_pc2_i, *ros_pc2_i, *ros_pc2_i));
        }
      }
    }
    // h2d
    cupoch_pc->points_.resize(cupoch_pc_points_host.size());
    cupoch_pc->colors_.resize(cupoch_pc_colors_host.size());

    // hipMemcpy to improve speed, Async to lower the cpu usage
    hipStream_t s0 = utility::GetStream(0);
    hipStream_t s1 = utility::GetStream(1);
    hipStreamAttachMemAsync(s0, thrust::raw_pointer_cast(cupoch_pc->points_.data()));
    cudaSafeCall(hipMemcpyAsync(thrust::raw_pointer_cast(cupoch_pc->points_.data()), cupoch_pc_points_host.data(),
                            cupoch_pc_points_host.size() * sizeof(Eigen::Vector3f), hipMemcpyHostToDevice, s0));
    hipStreamAttachMemAsync(s1, thrust::raw_pointer_cast(cupoch_pc->colors_.data()));
    cudaSafeCall(hipMemcpyAsync(thrust::raw_pointer_cast(cupoch_pc->colors_.data()), cupoch_pc_colors_host.data(),
                            cupoch_pc_colors_host.size() * sizeof(Eigen::Vector3f), hipMemcpyHostToDevice, s1));
    hipDeviceSynchronize();
  }

//   zs@zs-dell:~$ rostopic echo /velodyne_points
// header:
//   seq: 7302
//   stamp:
//     secs: 1607570362
//     nsecs: 406008720
//   frame_id: "velodyne"
// height: 1
// width: 25556
// fields:
//   -
//     name: "x"
//     offset: 0
//     datatype: 7
//     count: 1
//   -
//     name: "y"
//     offset: 4
//     datatype: 7
//     count: 1
//   -
//     name: "z"
//     offset: 8
//     datatype: 7
//     count: 1
//   -
//     name: "intensity"
//     offset: 12
//     datatype: 7
//     count: 1
//   -
//     name: "ring"
//     offset: 16
//     datatype: 4
//     count: 1
//   -
//     name: "time"
//     offset: 18
//     datatype: 7
//     count: 1
// is_bigendian: False
// point_step: 22
// row_step: 0
// data: [13, 38, 106, 192, 89

} // namespace cupoch_conversions
